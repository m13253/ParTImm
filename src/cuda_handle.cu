/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <unordered_map>
#include <ParTI/device.hpp>
#include <ParTI/errcode.hpp>
#include <ParTI/error.hpp>
#include <hipblas.h>
#include <hipsolver.h>
#include <hipsolver.h>

namespace pti {

namespace {

thread_local std::unordered_map<int, hipblasHandle_t> cublasHandles;
thread_local std::unordered_map<int, hipsolverHandle_t> cusolverDnHandles;
thread_local std::unordered_map<int, hipsolverSpHandle_t> cusolverSpHandles;

}

void* CudaDevice::GetCublasHandle() {
    try {
        return cublasHandles.at(cuda_device);
    } catch(std::out_of_range) {
        hipSetDevice(cuda_device);
        hipblasHandle_t handle = nullptr;
        hipblasStatus_t status = hipblasCreate(&handle);
        ptiCheckError(status != HIPBLAS_STATUS_SUCCESS, ERR_CUDA_LIBRARY, ("cuBLAS library error code " + std::to_string(status)).c_str());
        return handle;
    }
}

void* CudaDevice::GetCusolverDnHandle() {
    try {
        return cusolverDnHandles.at(cuda_device);
    } catch(std::out_of_range) {
        hipSetDevice(cuda_device);
        hipsolverHandle_t handle = nullptr;
        hipsolverStatus_t status = hipsolverDnCreate(&handle);
        ptiCheckError(status != HIPSOLVER_STATUS_SUCCESS, ERR_CUDA_LIBRARY, ("cuBLAS library error code " + std::to_string(status)).c_str());
        return handle;
    }
}

void* CudaDevice::GetCusolverSpHandle() {
    try {
        return cusolverSpHandles.at(cuda_device);
    } catch(std::out_of_range) {
        hipSetDevice(cuda_device);
        hipsolverSpHandle_t handle = nullptr;
        hipsolverStatus_t status = hipsolverSpCreate(&handle);
        ptiCheckError(status != HIPSOLVER_STATUS_SUCCESS, ERR_CUDA_LIBRARY, ("cuBLAS library error code " + std::to_string(status)).c_str());
        return handle;
    }
}

}
