/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <unordered_map>
#include <ParTI/device.hpp>
#include <ParTI/errcode.hpp>
#include <ParTI/error.hpp>
#include <ParTI/utils.hpp>
#include <hipblas.h>
#include <hipsolver.h>
#include <hipsolver.h>

namespace pti {

namespace {

thread_local std::unordered_map<int, hipblasHandle_t> cublasHandles;
thread_local std::unordered_map<int, hipsolverHandle_t> cusolverDnHandles;
thread_local std::unordered_map<int, hipsolverSpHandle_t> cusolverSpHandles;

thread_local struct HandleManager {
    template<typename T>
    T get(std::unordered_map<int, T>& m, int i) const {
        return m.at(i);
    }
    template<typename T>
    T set(std::unordered_map<int, T>& m, int i, T v) const {
        m.insert(std::pair<int, T>(i, v));
        return v;
    }
    ~HandleManager() {
        for(auto const& i : cublasHandles) {
            hipblasDestroy(i.second);
        }
        for(auto const& i : cusolverDnHandles) {
            hipsolverDnDestroy(i.second);
        }
        for(auto const& i : cusolverSpHandles) {
            hipsolverSpDestroy(i.second);
        }
    }
} manager;

}

void* CudaDevice::GetCublasHandle() {
    try {
        return manager.get(cublasHandles, cuda_device);
    } catch(std::out_of_range) {
        hipSetDevice(cuda_device);
        hipblasHandle_t handle = nullptr;
        hipblasStatus_t status = hipblasCreate(&handle);
        ptiCheckError(status != HIPBLAS_STATUS_SUCCESS, ERR_CUDA_LIBRARY, ("cuBLAS library error code " + std::to_string(status)).c_str());
        return manager.set(cublasHandles, cuda_device, handle);
    }
}

void* CudaDevice::GetCusolverDnHandle() {
    try {
        return manager.get(cusolverDnHandles, cuda_device);
    } catch(std::out_of_range) {
        hipSetDevice(cuda_device);
        hipsolverHandle_t handle = nullptr;
        hipsolverStatus_t status = hipsolverDnCreate(&handle);
        ptiCheckError(status != HIPSOLVER_STATUS_SUCCESS, ERR_CUDA_LIBRARY, ("cuBLAS library error code " + std::to_string(status)).c_str());
        return manager.set(cusolverDnHandles, cuda_device, handle);
    }
}

void* CudaDevice::GetCusolverSpHandle() {
    try {
        return manager.get(cusolverSpHandles, cuda_device);
    } catch(std::out_of_range) {
        hipSetDevice(cuda_device);
        hipsolverSpHandle_t handle = nullptr;
        hipsolverStatus_t status = hipsolverSpCreate(&handle);
        ptiCheckError(status != HIPSOLVER_STATUS_SUCCESS, ERR_CUDA_LIBRARY, ("cuBLAS library error code " + std::to_string(status)).c_str());
        return manager.set(cusolverSpHandles, cuda_device, handle);
    }
}

}
