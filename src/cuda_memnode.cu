#include "hip/hip_runtime.h"
/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI/memnode.hpp>
#include <ParTI/error.hpp>

namespace pti {

CudaMemNode::CudaMemNode(int cuda_device) {
    this->cuda_device = cuda_device;
}

void* CudaMemNode::malloc(size_t size) {
    hipError_t error;

    int old_device;
    error = hipGetDevice(&old_device);
    ptiCheckCUDAError(error);

    error = hipSetDevice(cuda_device);
    ptiCheckCUDAError(error);

    void* ptr;
    error = hipMalloc(&ptr, size);
    ptiCheckCUDAError(error);

    if(enable_profiling) {
        profile(ptr, size);
        std::fprintf(stderr, "[CudaMemNode] malloc(%zu),\t%s used, %s max\n", size, bytes_allocated_str().c_str(), max_bytes_allocated_str().c_str());
    }

    error = hipSetDevice(old_device);
    ptiCheckCUDAError(error);

    return ptr;
}

void* CudaMemNode::realloc(void*, size_t) {
    throw std::bad_alloc();
}

void CudaMemNode::free(void* ptr) {
    hipError_t error;

    int old_device;
    error = hipGetDevice(&old_device);
    ptiCheckCUDAError(error);

    error = hipSetDevice(cuda_device);
    ptiCheckCUDAError(error);

    error = hipFree(ptr);
    ptiCheckCUDAError(error);

    if(enable_profiling) {
        size_t oldsize = profile(ptr, 0);
        std::fprintf(stderr, "[CudaMemNode] free(%zu),\t%s used, %s max\n", oldsize, bytes_allocated_str().c_str(), max_bytes_allocated_str().c_str());
    }

    error = hipSetDevice(old_device);
    ptiCheckCUDAError(error);
}

void CudaMemNode::memcpy_to(void* dest, MemNode& dest_node, void* src, size_t size) {
    hipError_t error;
    if(CpuMemNode* cpu_dest_node = dynamic_cast<CpuMemNode*>(&dest_node)) {
        error = hipSetDevice(cuda_device);
        ptiCheckCUDAError(error);
        error = hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
        ptiCheckCUDAError(error);
    } else if(CudaMemNode* cuda_dest_node = dynamic_cast<CudaMemNode*>(&dest_node)) {
        error = hipMemcpyPeer(dest, cuda_dest_node->cuda_device, src, cuda_device, size);
        ptiCheckCUDAError(error);
    } else {
        ptiCheckError(true, 1, "Unknown memory node type");
    }
}

void CudaMemNode::memcpy_from(void* dest, void* src, MemNode& src_node, size_t size) {
    hipError_t error;
    if(CpuMemNode* cpu_src_node = dynamic_cast<CpuMemNode*>(&src_node)) {
        error = hipSetDevice(cuda_device);
        ptiCheckCUDAError(error);
        error = hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
        ptiCheckCUDAError(error);
    } else if(CudaMemNode* cuda_src_node = dynamic_cast<CudaMemNode*>(&src_node)) {
        error = hipMemcpyPeer(dest, cuda_device, src, cuda_src_node->cuda_device, size);
        ptiCheckCUDAError(error);
    } else {
        ptiCheckError(true, 1, "Unknown memory node type");
    }
}

}
