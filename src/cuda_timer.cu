/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI/timer.hpp>
#include <cstdio>
#include <ParTI/device.hpp>
#include <ParTI/error.hpp>
#include <ParTI/session.hpp>

namespace pti {

void Timer::cuda_init() {
    hipError_t error;

    error = hipEventCreate((hipEvent_t*) &cuda_start_event);
    ptiCheckCUDAError(error);

    error = hipEventCreate((hipEvent_t*) &cuda_stop_event);
    ptiCheckCUDAError(error);
}

void Timer::cuda_fini() {
    hipEventDestroy((hipEvent_t) cuda_start_event);
    hipEventDestroy((hipEvent_t) cuda_stop_event);
}

void Timer::cuda_start() {
    hipError_t error;

    int old_device;
    error = hipGetDevice(&old_device);
    ptiCheckCUDAError(error);

    error = hipSetDevice(cuda_dev->cuda_device);
    ptiCheckCUDAError(error);

    error = hipEventRecord((hipEvent_t) cuda_start_event);
    ptiCheckCUDAError(error);

    error = hipEventSynchronize((hipEvent_t) cuda_start_event);
    ptiCheckCUDAError(error);

    error = hipSetDevice(old_device);
    ptiCheckCUDAError(error);
}

void Timer::cuda_stop() {
    hipError_t error;

    int old_device;
    error = hipGetDevice(&old_device);
    ptiCheckCUDAError(error);

    error = hipSetDevice(cuda_dev->cuda_device);
    ptiCheckCUDAError(error);

    error = hipEventRecord((hipEvent_t) cuda_stop_event);
    ptiCheckCUDAError(error);

    error = hipEventSynchronize((hipEvent_t) cuda_stop_event);
    ptiCheckCUDAError(error);

    error = hipSetDevice(old_device);
    ptiCheckCUDAError(error);
}

double Timer::cuda_elapsed_time() const {
    float elapsed;
    if(hipEventElapsedTime(&elapsed, (hipEvent_t) cuda_start_event, (hipEvent_t) cuda_stop_event) != 0) {
        return NAN;
    }
    return elapsed * 1e-3;
}

}
