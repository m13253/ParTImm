/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <cstdio>
#include <ParTI/timer.hpp>
#include <ParTI/session.hpp>
#include <ParTI/device.hpp>

namespace pti {

void Timer::cuda_init() {
    hipEventCreate((hipEvent_t*) &cuda_start_event);
    hipEventCreate((hipEvent_t*) &cuda_stop_event);
}

void Timer::cuda_fini() {
    hipEventDestroy((hipEvent_t) cuda_start_event);
    hipEventDestroy((hipEvent_t) cuda_stop_event);
}

void Timer::cuda_start() {
    hipSetDevice(cuda_dev->cuda_device);
    hipEventRecord((hipEvent_t) cuda_start_event);
    hipEventSynchronize((hipEvent_t) cuda_start_event);
}

void Timer::cuda_stop() {
    hipSetDevice(cuda_dev->cuda_device);
    hipEventRecord((hipEvent_t) cuda_stop_event);
    hipEventSynchronize((hipEvent_t) cuda_stop_event);
}

double Timer::elapsed_time() const {
    float elapsed;
    if(hipEventElapsedTime(&elapsed, (hipEvent_t) cuda_start_event, (hipEvent_t) cuda_stop_event) != 0) {
        return NAN;
    }
    return elapsed * 1e-3;
}

}
